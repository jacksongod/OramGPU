#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "aesctr.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);




__global__ void aeskernel(aes_context *ctx,
                     //  size_t length,
                     //  size_t *nc_off,
                    //   unsigned char nonce_counter[16],
                    //   unsigned char stream_block[16],
                       const unsigned char *input,
                       unsigned char *output )
{
    int i = threadIdx.x;

}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
