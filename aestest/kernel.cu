#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "aesctr.h"
#include <cstring>
#include <iostream>
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
printf("Error at %s:%d\n",__FILE__,__LINE__); \
return EXIT_FAILURE;}} while(0)
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);


__device__ void aes_fround (int idx, int whichword,int laneid, int& outword, int& inword,uint32_t RK){
	int secondposition = (whichword+1)%4-whichword+laneid;
	int thirdposition = (whichword+2)%4-whichword+laneid; 
	int fourthposition = (whichword+3)%4-whichword+laneid;
	int ysecond = __shfl(inword, secondposition);
	int ythird = __shfl(inword,thirdposition);
	int yfourth = __shfl(inword, fourthposition);

	outword = RK^DFT0[ ( inword   ) & 0xFF ] ^ 
		         DFT1[ ( ysecond >>  8 ) & 0xFF ] ^
                 DFT2[ ( ythird >> 16 ) & 0xFF ] ^   
                 DFT3[ ( yfourth >> 24 ) & 0xFF ];    


}
__device__ void aes_finalfround (int idx, int whichword,int laneid, int& outword, int& inword,uint32_t RK){
	int secondposition = (whichword+1)%4-whichword+laneid;
	int thirdposition = (whichword+2)%4-whichword+laneid; 
	int fourthposition = (whichword+3)%4-whichword+laneid;
	int ysecond = __shfl(inword, secondposition);
	int ythird = __shfl(inword,thirdposition);
	int yfourth = __shfl(inword, fourthposition);

	outword =RK ^ ( (uint32_t) DFSb[ ( inword       ) & 0xFF ]       ) ^
                ( (uint32_t) DFSb[ ( ysecond >>  8 ) & 0xFF ] <<  8 ) ^
                ( (uint32_t) DFSb[ ( ythird >> 16 ) & 0xFF ] << 16 ) ^
                ( (uint32_t) DFSb[ ( yfourth >> 24 ) & 0xFF ] << 24 );  

                

}

__global__ void aeskernel(aes_context *ctx,
                     //  size_t length,
                     //  size_t *nc_off,
                    //   unsigned char nonce_counter[16],
                    //   unsigned char stream_block[16],
                       const unsigned char *nonce_counter,
                      // unsigned char **stream_block,
					  // unsigned char **input,
					   unsigned char *inout,
					   size_t pitch_nc,
					   size_t pitch_io
					   )
{
    int idx = threadIdx.x;
	int xword, yword; 
	int whichoramblock = idx/16; 
	int whichcipherblock = idx/4%4; 
	int laneid = idx%32;
	int whichword = idx%4;
    uint32_t* RK_ptr = ctx->rk;

	GET_UINT32_LE( xword, &nonce_counter[whichoramblock*pitch_nc+whichcipherblock*16+whichword*4],  whichword*4 ); 
	xword ^= *(RK_ptr+whichword);
	RK_ptr += 4; 
	for( int i = (ctx->nr >> 1) - 1; i > 0; i-- )
        {
            aes_fround( idx, whichword, laneid, yword, xword,*(RK_ptr+whichword));
			RK_ptr += 4; 
            aes_fround( idx, whichword, laneid, xword, yword,*(RK_ptr+whichword));
			RK_ptr += 4; 
        }

        aes_fround( idx, whichword, laneid, yword, xword,*(RK_ptr+whichword) );
		RK_ptr += 4; 
		aes_finalfround ( idx, whichword, laneid, xword, yword,*(RK_ptr+whichword) );

//		PUT_UINT32_LE( xword, &stream_block[whichoramblock][whichcipherblock*16+whichword*4],  whichword*4);
		*((uint32_t*)&inout[whichoramblock*pitch_io+whichcipherblock*16+whichword*4]) = 
			*((uint32_t*)&inout[whichoramblock*pitch_io+whichcipherblock*16+whichword*4])^
			xword; 
			//*((uint32_t*)&stream_block[whichoramblock][whichcipherblock*16+whichword*4]);

}

int main()
{
    // int i, j, u, v;
    unsigned char key[16];
    
    unsigned char iv[16];


    size_t offset;

    int len;
	int height = 24;     //num of oram blocks; 
	int width = 64;      //num of bytes in one oram block
    unsigned char* nonce_counter = new unsigned char[height*width];
  //  unsigned char* stream_block= new unsigned char[height*width];
	unsigned char* buf= new unsigned char[height*width];
    aes_context ctx;
	 len = 16;
	for ( int i = 0; i< height; i++){

		for (int j = 0; j<width/16; j++){
			memcpy( &nonce_counter[64*i+j*16], aes_test_ctr_nonce_counter[0], 16 );
			memcpy( &buf[64*i+j*16], aes_test_ctr_ct[0], len );
		}
	}
    memcpy( key, aes_test_ctr_key[0], 16 );
	 offset = 0;
     aes_context::aes_setkey_enc( &ctx, key);

	size_t pitch_nc, pitch_buf,pitch_sb;
	unsigned char* dnonce_counter; 
	unsigned char* dbuf ; 
	aes_context* dctx;

	//allocate device memory 
	//unsigned char* stream_block;
	CUDA_CALL(hipMalloc(&dctx, sizeof(aes_context)));
	CUDA_CALL(hipMallocPitch(&dnonce_counter,&pitch_nc, width, height));
	CUDA_CALL(hipMallocPitch(&dbuf, &pitch_buf, width, height));

	//copy data to device memory 
	CUDA_CALL(hipMemcpy2D(dnonce_counter,pitch_nc,buf,width,width,height,hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy2D(dbuf,pitch_buf,buf,width,width,height,hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(dctx,&ctx,sizeof(aes_context),hipMemcpyHostToDevice));
   // hipMallocPitch(&stream_block, &pitch_sb, sizeof(float)*width, height);

	//copy constant data to device memory 
	 hipMemcpyToSymbol(HIP_SYMBOL(  DFT0),  FT0,   sizeof(uint32_t)*256  );
	 hipMemcpyToSymbol(HIP_SYMBOL(  DFT1),  FT1,   sizeof(uint32_t)*256  );
	 hipMemcpyToSymbol(HIP_SYMBOL(  DFT2),  FT2,   sizeof(uint32_t)*256  );
	 hipMemcpyToSymbol(HIP_SYMBOL(  DFT3),  FT3,   sizeof(uint32_t)*256  );
	 hipMemcpyToSymbol(HIP_SYMBOL(  DFSb),  FSb,   sizeof(char)*256  );
	aeskernel<<<1,height*width>>>(dctx,dnonce_counter,dbuf,pitch_nc,pitch_buf);
    CUDA_CALL(hipMemcpy2D(buf,width,dbuf,pitch_buf,width,height,hipMemcpyDeviceToHost));
    // Add vectors in parallel.
	
 
    CUDA_CALL( hipDeviceReset());

	delete[] buf; 
	delete[] nonce_counter;
	
    return 0;
}
