#include "hip/hip_runtime.h"

// Ensure printing of CUDA runtime errors to console (define before including cub.h)
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include "bucket.h"
#include <hiprand/hiprand_kernel.h>

#define CUDATHREADNUMLOG 10
#define CUDATHREADNUM (1<<CUDATHREADNUMLOG)


#define BLOCKNUMLOG 12
#define MAPSIZEPERTHREAD 4
#define BLOCKSIZELOG 12
#define LEAFNUMLOG  11
#define TREESIZE (1<<LEAFNUMLOG)*2//-1
#define CUDABLOCKNUM 1
#define BLOCKPERBUCKET 2

#define STASHSIZE 256

#define ACCESSNUM 1000
/**
 * Main
 */
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
printf("Error at %s:%d\n",__FILE__,__LINE__); \
return EXIT_FAILURE;}} while(0)

typedef TBucket<2> OramB; 

__device__ __forceinline__ int calcindex(int level, uint16_t path){
    return (path>>(LEAFNUMLOG-level))+ (1<<level) -1 ;

}

__global__ void oramshare(uint16_t* position_table, uint32_t* access_script,uint16_t* checktable, OramB* oramtree, uint16_t* checktable2,hiprandState *randstate){
	int tid =  threadIdx.x;
        hiprandState localrandState = randstate[tid];
        __shared__ OramB metatree[TREESIZE];              //4K*sizeof(OramB) = 16KB
       // __shared__ OramB treepath[LEAFNUMLOG+1];          //12*sizeof(OramB) = 48B
        __shared__ uint32_t treepathlock[(LEAFNUMLOG+1)*2];    //96B
        __shared__ uint32_t streepathlock[STASHSIZE];    //1kB
        __shared__ uint16_t stash [STASHSIZE];            //256B
        __shared__ uint32_t stashlock[STASHSIZE];          //1kB
	__shared__ uint16_t localtable[1<<(BLOCKNUMLOG)];   //8KB  when blocknumlog = 12
        __shared__  uint16_t newposition; 
        __shared__ uint32_t stashcount;
        __shared__ uint32_t pathcount;
        __shared__ uint32_t stashaccessloc;
                                                          //total <42 KB
       
    //copy position table from global memory to shared memory
       localtable[tid*MAPSIZEPERTHREAD] = position_table[tid*MAPSIZEPERTHREAD];
       localtable[tid*MAPSIZEPERTHREAD+1] = position_table[tid*MAPSIZEPERTHREAD+1];
       localtable[tid*MAPSIZEPERTHREAD+2] = position_table[tid*MAPSIZEPERTHREAD+2];
       localtable[tid*MAPSIZEPERTHREAD+3] = position_table[tid*MAPSIZEPERTHREAD+3];
     //localtable[tid*2] = (position_table[tid*MAPSIZEPERTHREAD+1]<<16) | position_table[tid*MAPSIZEPERTHREAD];
     //localtable[tid*2+1] = (position_table[tid*MAPSIZEPERTHREAD+3]<<16) | position_table[tid*MAPSIZEPERTHREAD+2];
 
    //memcpy(&localtable[tid*MAPSIZEPERTHREAD],&position_table[tid*MAPSIZEPERTHREAD],sizeof(uint16_t)*MAPSIZEPERTHREAD);
    // copy metadata tree from global memory to shared memory
    memcpy(&metatree[tid*4], &oramtree[tid*4],sizeof(OramB)*4);
   if (tid <256)  stashlock[tid] = 0;
  // if (tid< 24) treepathlock[tid] = 0;
  // if (tid<12) streepathlock[tid] = 0;
   //if (tid ==256) pathcount = 24;
   if (tid ==512) stashcount = STASHSIZE;
  //  if (tid< (LEAFNUMLOG+1)){
  //  memset(&treepathlock[tid],0x0,4);
    
  //  }//else if (tid==1023){
  //   stashcount =0;
  //   pathcount = 0;
  //  }
   __syncthreads();
    
                       //checktable[tid] = 1;
    uint32_t pathid; 
    //uint32_t pathidtemp;
    //uint32_t accessid; 
    //bool r_foundposition=false;
    //bool w_foundposition=false;; 
    int startindex= tid<<3 ; //tid *8
    uint16_t blockid;
	for (int i = 0 ; i<ACCESSNUM ; i++ ){
               blockid = access_script[i];
               pathid = localtable[blockid];
               if ((blockid>>3) == (tid-512)){
                   newposition =(hiprand(&localrandState))& 0x7ff; 
               }
            //    accessid = access_script[i];
	    //	pathidtemp = localtable[accessid/2] ;
            //    pathid =   0xffff&(pathidtemp  >>((accessid&0x1)<<4)); 
                    
        if(tid <256)  streepathlock[tid] = 0;
         
        
         if (tid< 24){  //copy entire path to local registers (12 levels, 24 blocks)
               treepathlock[tid] = 0;
                //pathcount = 24; 
            //printf("rand : %d is %d\n",tid,  (unsigned)(hiprand(&localrandState))%(1<<LEAFNUMLOG));
           int treeindex = calcindex(tid/2, pathid);
           //printf("id: %d, index %d\t",tid,treeindex );
           uint16_t id = metatree[treeindex].id[tid%2];
           if((id>>15) == 1){   // if data is valid 
            // printf("id: %d valid data \t ", tid); 
              while(true){
                 if (!atomicCAS(&stashlock[startindex],0,1 )){
            //         printf("id: %d, foundposition\t",tid);
                    stash[startindex++] = id &0x7ff; 
                    //atomicSub(&stashcount,1);
              //       printf("id: %d, data id %d\n",tid,stash[startindex%STASHSIZE]);
              //      checktable[i*24+tid] = stash[startindex] ;
              //      checktable2[i*24+tid] = pathid;
              //      printf("after id %d, checktable data %d \n", tid, checktable[i*24+tid]);
                    break; 
                 }
                 startindex = (startindex+1)%STASHSIZE; 
              }
            //   printf("out\n");
           }  
                     
         }  
         __syncthreads();
         
        /* if (tid < STASHSIZE){
             if (stashlock[tid]!=0){
		   int myblockid = stash[tid];
		   if(myblockid == blockid ){
                      localtable[myblockid] = newposition;
                      stashaccessloc = tid;
		   }
                      
                   int sortkey = localtable[myblockid] ^ pathid;  
                   int level = __clz((sortkey<<21)|0x00100000); 
                   int treeindex = calcindex(level,pathid);
                while(true){
                   if (pathcount<=0) break; 
                   if(!atomicCAS(&treepathlock[level<<1],0,1)){
                       metatree[treeindex].id[0] = myblockid;
                       stashlock[tid] = 0;
                       atomicAdd(&stashcount,1);
                       atomicSub(&pathcount,1);
                       break; 
                   } else if (!atomicCAS(&treepathlock[(level<<1)+1],0,1)){
                       metatree[treeindex].id[1] = myblockid;
                       stashlock[tid] = 0;
                       atomicAdd(&stashcount,1);
                       atomicSub(&pathcount,1);
                       break;
                   } 
                   level--; 
                   if (level<0) break;   
                   treeindex = (treeindex-1)>>1;
                    
                   
                }		       

             }


         } */ 
         //if (tid <256)  stashlock[tid] = 0;      
         if (tid < STASHSIZE*2){
              int secondblock = (tid>=STASHSIZE);
              int stid = tid-secondblock*STASHSIZE; 
             if (stashlock[stid]!=0){
		   int myblockid = stash[stid] ;
		   if(myblockid == blockid ){
                      localtable[myblockid] = newposition;
                      stashaccessloc = stid;
		   }
                      
                   int sortkey = localtable[myblockid] ^ pathid;  
                   int level = __clz((sortkey<<21)|0x00100000); 
                   int treeindex = calcindex(level,pathid);
                while(true){
                 //  if (pathcount<=0) break; 
                   if(!atomicCAS(&treepathlock[(level<<1)+secondblock],0,1)){
                       if (atomicCAS(&streepathlock[stid],0,1)) {
                          treepathlock[(level<<1)+secondblock] = 0;
                          break;
                       } 
                       metatree[treeindex].id[secondblock] = myblockid;
                       stashlock[stid] = 0;
                   //    atomicAdd(&stashcount,1);
                   //    atomicSub(&pathcount,1);
                       break; 
                   } 
                   level--; 
                   if (level<0) break;   
                   treeindex = (treeindex-1)>>1;
                    
                   
                }		       

             }


         }  
         __syncthreads();

	}

        
}
__global__ void setup_kernel(hiprandState *state)
{
int id = threadIdx.x;
/* Each thread gets same seed, a different sequence number,
no offset */
hiprand_init(1234, id, 0, &state[id]);
}

int main(int argc, char** argv)
{
    // Initialize command line
	 hipDeviceSetCacheConfig(hipFuncCachePreferShared);
//	 hipDeviceSetCacheConfig(hipFuncCachePreferL1);
      
    printf("start\n");
   hiprandState *devStates;
   CUDA_CALL(hipMalloc((void **)&devStates, 1024 * sizeof(hiprandState)));
   setup_kernel<<<CUDABLOCKNUM,CUDATHREADNUM>>>(devStates);
    uint16_t* p_table = new uint16_t[1<<BLOCKNUMLOG];
    uint16_t* check_table = new uint16_t[ACCESSNUM*24];
    uint16_t* check_table2 = new uint16_t[ACCESSNUM*24];
    uint32_t* access_script = new uint32_t[ACCESSNUM];
    //uint32_t* orampath = new uint32_t[1<<LEAFNUMLOG];
    OramB* oramtree = new OramB[TREESIZE];
    for (int i = 0; i< (1<<(BLOCKNUMLOG)); i++){
        p_table[i] = 0xdead;
    }
    printf("finished initialize raw p_table\n");
    for (int i = 0; i< (TREESIZE-1); i++){
            int temp = i;
            while (temp <(TREESIZE-1)){
                temp = temp*2 + 1+ 1*(temp%2);
            }
             temp = (temp-1)/2 + 1 - (1<<LEAFNUMLOG);
        oramtree[i].id[0] =( 0x8000 |rand()%(1<<BLOCKNUMLOG)); 
        p_table[oramtree[i].id[0]%(1<<BLOCKNUMLOG)] = temp; 
        oramtree[i].id[1] =( 0x8000| rand()%(1<<BLOCKNUMLOG)); 
        p_table[oramtree[i].id[1]%(1<<BLOCKNUMLOG)] = temp; 
    }
    printf ("finished initializa p_table \n");
    printf ("Accessing %d blocks \n", ACCESSNUM);
    /*for (int i = 0; i< (1<<(BLOCKNUMLOG)); i++){
        bool find = false; 
        for (int j = 0; j< (TREESIZE-1); j++){
            int temp = j;
            while (temp <TREESIZE-1){
                temp = temp*2 + 1*(temp%2);
            }
             temp = (temp-1)/2 + 1 - (1<<LEAFNUMLOG);
            if(i == oramtree[j].id[0]){
               oramtree[j].id[0] = 0x8000 | (oramtree[j].id[0]) ; 
               p_table[i] = temp;
               find = true;
               break; 
            } else if (i == oramtree[j].id[1]){
               oramtree[j].id[1] = 0x8000 | (oramtree[j].id[1]) ; 
               p_table[i] = temp;
               find = true;
               break;
            } 
            
        }
        if (!find) {
            p_table[i] = 0xdead;
            printf("not able to find %d\n", i);
        } 
    }*/
    for (int i = 0; i<(ACCESSNUM); i++){
        do {
        access_script[i] = rand()%(1<<BLOCKNUMLOG);
        } while(p_table[access_script[i]] == 0xdead); 
        int bucketindex = (1<<LEAFNUMLOG)-1+p_table[access_script[i]];
        while (true){
            if(oramtree[bucketindex].id[0] == (access_script[i] | 0x8000) || oramtree[bucketindex].id[1] == (access_script[i] | 0x8000)){
               break;
            }
            if (bucketindex <= 0) {
               printf( "fail: cant find blockid in tree: %d\n", access_script[i]); 
               break;
            }
            bucketindex=(bucketindex-1 )/2;   

        }
       // printf("host access : 0x%x\n", p_table[access_script[i]] );
    }
    printf("finish initialing host\n");
    printf("orambucket size %d \n",sizeof(OramB));
    uint16_t* cup_table;
    uint16_t* cucheck_table;
    uint16_t* cucheck_table2;
    uint32_t* cuaccess_script;
   // uint32_t* cuorampath;
    OramB* cuoramtree;
    
    hipError_t pterr = hipMalloc((void**)&cup_table,sizeof(uint16_t) *( 1<<BLOCKNUMLOG));
    if(pterr != hipSuccess){
     printf("The pterror is %s", hipGetErrorString(pterr));
    }
    hipError_t err = hipMalloc((void**)&cucheck_table,sizeof(uint16_t)*(ACCESSNUM*24));
    if(err != hipSuccess){
     printf("The error is %s", hipGetErrorString(err));
    }
    hipError_t errr = hipMalloc((void**)&cucheck_table2,sizeof(uint16_t)*(ACCESSNUM*24));
    if(errr != hipSuccess){
     printf("The error2 is %s", hipGetErrorString(errr));
    }
    hipMalloc((void**)&cuaccess_script,sizeof(uint32_t) *(ACCESSNUM));
   // hipMalloc((void**)&cuorampath,sizeof(uint32_t) *( 1<<LEAFNUMLOG));
    hipMalloc((void**)&cuoramtree,sizeof(OramB) *( TREESIZE));
    
    hipError_t pterr2 = hipMemcpy(cup_table, p_table, (1<<BLOCKNUMLOG) * sizeof(uint16_t),hipMemcpyHostToDevice);
    if(pterr2 != hipSuccess){
     printf("The pt copy htom error is %s", hipGetErrorString(pterr2));
    }

    hipMemcpy(cuaccess_script, access_script, (ACCESSNUM) * sizeof(uint32_t),hipMemcpyHostToDevice);
   // hipMemcpy(cuorampath, orampath, (1<<LEAFNUMLOG) * sizeof(uint32_t),hipMemcpyHostToDevice);
    hipMemcpy(cuoramtree, oramtree, (TREESIZE) * sizeof(OramB),hipMemcpyHostToDevice);
   
    oramshare<<<CUDABLOCKNUM,CUDATHREADNUM>>>(cup_table,cuaccess_script,cucheck_table, cuoramtree, cucheck_table2, devStates);
    if (hipPeekAtLastError() != hipSuccess) {
    	printf("The peek last error is %s", hipGetErrorString(hipGetLastError()));
    }
    hipDeviceSynchronize();
    
    hipError_t err2 = hipMemcpy(check_table, cucheck_table, (ACCESSNUM*24) * sizeof(uint16_t), hipMemcpyDeviceToHost);
    if(err2 != hipSuccess){
     printf("after  checktable copy error is %s\n", hipGetErrorString(err2));
    }
    hipError_t err3 = hipMemcpy(check_table2, cucheck_table2, (ACCESSNUM*24) * sizeof(uint16_t), hipMemcpyDeviceToHost);
    if(err3 != hipSuccess){
     printf("after  checktable copy error is %s\n", hipGetErrorString(err3));
    }
    printf("gpu finished\n");
  /*  bool pass = true; 
    for (int i =0 ; i< ACCESSNUM ; i++){
       int bucketindex = (1<<LEAFNUMLOG) - 1 + p_table[access_script[i]]; 
       //printf ("bucket index %d \n", bucketindex);
       for(int j = 11 ; j >= 0; j--) {
         
        if (check_table[i*24+j*2] !=  oramtree[bucketindex].id[0]){
            pass = false; 
            printf("fail 0 id: 0x%x 0x%x real id 0x%X,  0x%x\n" ,bucketindex,oramtree[bucketindex].id[0],check_table2[i*24+j*2], check_table[i*24+j*2] );
        }
        else
        { 
            printf("pass 0 id: 0x%x 0x%x real id 0x%x, 0x%x\n" ,bucketindex,oramtree[bucketindex].id[0],check_table2[i*24+j*2] ,check_table[i*24+j*2] );
        }
        if (check_table[i*24+j*2+1] !=  oramtree[bucketindex].id[1]){
            pass = false;
            printf("fail 1 id: 0x%x 0x%x real id 0x%x 0x%x\n" ,bucketindex, oramtree[bucketindex].id[1], check_table2[i*24+j*2+1],check_table[i*24+j*2+1] );
           
        }
        else
        { 
            printf("pass 0 id: 0x%x 0x%x real id 0x%x 0x%x\n" ,bucketindex,oramtree[bucketindex].id[0],check_table2[i*24+j*2+1] ,check_table[i*24+j*2] );
        }
         bucketindex = (bucketindex-1)/2; 

       }
    }
    
    printf("\nfinished \n");
    if (pass) {
		printf("All data correct\n");
	}else{
		printf("Some data not correct\n");
	}
    */
    hipFree(cuaccess_script);
    hipFree(cup_table);
    hipFree(cucheck_table);
    hipFree(cucheck_table2);
    //hipFree(cuorampath);
    hipFree(cuoramtree);
    hipFree(devStates);
    delete[] p_table;
    delete[] access_script;
    //delete[] orampath;
    delete[] check_table; 
    delete[] check_table2; 
    delete[] oramtree; 
    hipDeviceReset();
    return 0;
}
