#include "hip/hip_runtime.h"

// Ensure printing of CUDA runtime errors to console (define before including cub.h)
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include "bucket.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "atomicfifo.h"

#define CUDATHREADNUMLOG 9
#define CUDATHREADNUM (1<<CUDATHREADNUMLOG)


#define BLOCKNUMLOG 12
#define MAPSIZEPERTHREAD 8
#define BLOCKSIZE 64
#define LEAFNUMLOG  11
#define TREESIZE (1<<LEAFNUMLOG)*2//-1
#define CUDABLOCKNUM 1
#define BLOCKPERBUCKET 2

#define STASHSIZE 256

#define ACCESSNUM 1000
/**
 * Main
 */
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
printf("Error at %s:%d\n",__FILE__,__LINE__); \
return EXIT_FAILURE;}} while(0)

typedef TBucket<BLOCKPERBUCKET> OramB; 
typedef TDBucket<BLOCKPERBUCKET,BLOCKSIZE> OramD;



__device__ __forceinline__ int calcindex(int level, uint16_t path){
    return (path>>(LEAFNUMLOG-level))+ (1<<level) -1 ;

}

__global__ void oramshare(uint16_t* position_table, uint32_t* access_script,uint16_t* checktable, OramB* oramtree, TDBlock<BLOCKSIZE>* checktable2,hiprandState *randstate,OramD* datatree){
	int tid =  threadIdx.x;
        hiprandState localrandState = randstate[tid];
        __shared__ OramB metatree[TREESIZE];              //4K*sizeof(OramB) = 16KB
       // __shared__ OramB treepath[LEAFNUMLOG+1];          //12*sizeof(OramB) = 48B
        __shared__ uint32_t treepathlock[(LEAFNUMLOG+1)*2];    //96B
        __shared__ uint32_t streepathlock[STASHSIZE];    //1kB
        __shared__ uint16_t stash [STASHSIZE];            //256B
        __shared__ uint32_t expectedblockindex; 
   //     __shared__ uint32_t stashlock[STASHSIZE];          //1kB
        __shared__ uint32_t camfifo[STASHSIZE]; 
        __shared__ int32_t front,end;
        __shared__ uint32_t  mutex; 
	__shared__ uint16_t localtable[1<<(BLOCKNUMLOG)];   //8KB  when blocknumlog = 12
        __shared__  uint16_t newposition; 
        __shared__ uint32_t stashcount;
		__shared__ uint32_t maxstashcount;
        __shared__ uint32_t pathcount;
        __shared__ uint32_t stashaccessloc[(LEAFNUMLOG+1)*BLOCKPERBUCKET]; //2*12*4B = 96B
        __shared__ uint32_t writebackloc[(LEAFNUMLOG+1)*BLOCKPERBUCKET];   //2*12*4B = 96B
        __shared__ uint32_t  datastash[STASHSIZE*(BLOCKSIZE/4)];          //4B * 256*16 = 16KB
		//__shared__ TDBlock<BLOCKSIZE> garbage_collector; 
		//__shared__ uint32_t blockinstash;  
       
    //copy position table from global memory to shared memory
     //  localtable[tid*MAPSIZEPERTHREAD] = position_table[tid*MAPSIZEPERTHREAD];
     //  localtable[tid*MAPSIZEPERTHREAD+1] = position_table[tid*MAPSIZEPERTHREAD+1];
     //  localtable[tid*MAPSIZEPERTHREAD+2] = position_table[tid*MAPSIZEPERTHREAD+2];
     //  localtable[tid*MAPSIZEPERTHREAD+3] = position_table[tid*MAPSIZEPERTHREAD+3];
     //localtable[tid*2] = (position_table[tid*MAPSIZEPERTHREAD+1]<<16) | position_table[tid*MAPSIZEPERTHREAD];
     //localtable[tid*2+1] = (position_table[tid*MAPSIZEPERTHREAD+3]<<16) | position_table[tid*MAPSIZEPERTHREAD+2];
 
    memcpy(&localtable[tid*MAPSIZEPERTHREAD],&position_table[tid*MAPSIZEPERTHREAD],sizeof(uint16_t)*MAPSIZEPERTHREAD);
    // copy metadata tree from global memory to shared memory
    memcpy(&metatree[tid*MAPSIZEPERTHREAD], &oramtree[tid*MAPSIZEPERTHREAD],sizeof(OramB)*MAPSIZEPERTHREAD);
   if (tid <256) {
	// stashlock[tid] = 0;
       camfifo[tid] = tid; 
       stash[tid] = 0xffff; 
   }
  // if (tid< 24) treepathlock[tid] = 0;
  // if (tid<12) streepathlock[tid] = 0;
   //if (tid ==256) pathcount = 24;
   if (tid ==511) {
	stashcount = STASHSIZE;
         end = mutex = 0; 
        front = STASHSIZE -1; 
   }
   if(tid == 384) maxstashcount = 256;
   
  //  if (tid< (LEAFNUMLOG+1)){
  //  memset(&treepathlock[tid],0x0,4);
    
  //  }//else if (tid==1023){
  //   stashcount =0;
  //   pathcount = 0;
  //  }
   __syncthreads();
    
                       //checktable[tid] = 1;
    uint32_t pathid; 
    //uint32_t pathidtemp;
    //uint32_t accessid; 
    //bool r_foundposition=false;
    //bool w_foundposition=false;; 
   // int startindex= tid/2 ; //tid *8
    uint16_t blockid;
	for (int i = 0 ; i<ACCESSNUM ; i++ ){
               blockid = access_script[i];
               pathid = localtable[blockid];
         //      if ((blockid>>3) == (tid-512)){
                   newposition =(hiprand(&localrandState))& 0x7ff; 
          //     }
            //    accessid = access_script[i];
	    //	pathidtemp = localtable[accessid/2] ;
            //    pathid =   0xffff&(pathidtemp  >>((accessid&0x1)<<4)); 
  //       int myblockid ;
       // if(tid <STASHSIZE)  {
	//		streepathlock[tid] = 0;}
		//	 myblockid = stash[tid] ;
		//	if(blockid == myblockid &&stashlock[tid] !=0){
		//		blockinstash = 1; 
		//		checktable[i] = blockid;
		//	} else{
		//		blockinstash =0; 
		//	}
		//} 
  //      __syncthreads();

		//if (!blockinstash) {
         if (tid< 384 && tid%16==0){  //copy entire path to local registers (12 levels, 24 blocks)
               int stid = tid /16;
               treepathlock[stid] = 0;
                //pathcount = 24; 
            //printf("rand : %d is %d\n",tid,  (unsigned)(hiprand(&localrandState))%(1<<LEAFNUMLOG));
           int treeindex = calcindex(stid/2, pathid);
           //printf("id: %d, index %d\t",tid,treeindex );
           uint16_t id = metatree[treeindex].id[stid%2];
		   metatree[treeindex].id[stid%2] = 0;
           if((id>>15) == 1){   // if data is valid 
            // printf("id: %d valid data \t ", tid); 
              /*while(true){
                 if (!atomicCAS(&stashlock[startindex],0,1 )){
            //         printf("id: %d, foundposition\t",tid);
                    stashaccessloc[stid] = startindex;
                    stash[startindex] = id &0xfff; 
                    
                 startindex = (startindex+1)%STASHSIZE; 
                  //  atomicSub(&stashcount,1);
              //       printf("id: %d, data id %d\n",tid,stash[startindex%STASHSIZE]);
              //      checktable[i*24+tid] = stash[startindex] ;
              //      checktable2[i*24+tid] = pathid;
              //      printf("after id %d, checktable data %d \n", tid, checktable[i*24+tid]);
                    break; 
                 }
                 startindex = (startindex+1)%STASHSIZE; 
	       } */
            //   printf("out\n");
               bool needlock = true; 
               int positionindex ; 
                       while(needlock) { 
                       	if (getlockattempt(&mutex)) {
                                positionindex = fifopop(camfifo, &front,&end);
                       		releaselock(&mutex);
                                needlock = false; 
                        }

                       }
              stashaccessloc[stid] = positionindex;
              stash[positionindex] = id & 0xfff;
		 }else{
		   stashaccessloc[stid] = 999;
         	   }
		   
                     
         }  
         __syncthreads();
        // if (tid==0 ) atomicMin(&maxstashcount, stashcount);
	//	 if(i==80){

	//		 int myball = 1000;
	//	 }
		
         if (tid < STASHSIZE){
             if (stash[tid]!=0xffff){
		   int myblockid = stash[tid];
		   if(myblockid == blockid ){
                      localtable[myblockid] = newposition;
                      checktable[i] = blockid;
                      expectedblockindex = tid;
		   }
                      
                   int sortkey = localtable[myblockid] ^ pathid;  
                   int level = __clz((sortkey<<21)|0x00100000); 
                   int treeindex = calcindex(level,pathid);
                while(true){
				   int blockloc = (level<<1);
                   if(!atomicCAS(&treepathlock[level<<1],0,1)){
                       writebackloc[blockloc] = tid; 
                       metatree[treeindex].id[0] = 0x8000|myblockid;
                       bool needlock = true; 
                       while(needlock) { 
                       	if (getlockattempt(&mutex)) {
                       		fifopush(camfifo,&front,&end,tid);
                       		releaselock(&mutex);
                                needlock = false; 
                        }

                       }
                      stash[tid] = 0xffff;
                      // stashlock[tid] = 0;
                //       atomicAdd(&stashcount,1);
                    //   atomicSub(&pathcount,1);
                       break; 
                   } else if (!atomicCAS(&treepathlock[(level<<1)+1],0,1)){
                       writebackloc[blockloc+1] = tid; 
                       metatree[treeindex].id[1] = 0x8000|myblockid;
                       bool needlock = true; 
                       while(needlock) { 
                       	if (getlockattempt(&mutex)) {
                       		fifopush(camfifo,&front,&end,tid);
                       		releaselock(&mutex);
                                needlock = false; 
                        }

                       }
                      stash[tid] = 0xffff;
                      // stashlock[tid] = 0;
              //         atomicAdd(&stashcount,1);
                     //  atomicSub(&pathcount,1);
                       break;
                   } 
                   level--; 
                   if (level<0) break;   
                   treeindex = (treeindex-1)>>1;
                    
                   
                }		       

             }


         } 
         //if (tid <256)  stashlock[tid] = 0;      
      /*   if (tid < STASHSIZE*BLOCKPERBUCKET){
              bool secondblock = (tid>=STASHSIZE);
              int stid = tid-secondblock*STASHSIZE; 
             if (stashlock[stid]!=0){
		   int myblockid = stash[stid] ;
		   if(myblockid == blockid ){
                      localtable[myblockid] = newposition;
					   checktable[i] = blockid;
                      expectedblockindex = stid;
                      //stashaccessloc = stid;
		   }
                      
                   int sortkey = localtable[myblockid] ^ pathid;  
                   int level = __clz((sortkey<<21)|0x00100000); 
                   int treeindex = calcindex(level,pathid);
                while(true){
                 //  if (pathcount<=0) break;
                   int blockloc = (level<<1) +secondblock;
                   if(!atomicCAS(&treepathlock[blockloc],0,1)){
                       if (atomicCAS(&streepathlock[stid],0,1)) {
                          treepathlock[blockloc] = 0;
                          break;
                       } 
                       writebackloc[blockloc] = stid; 
                       metatree[treeindex].id[secondblock] = 0x8000|myblockid;
                       stashlock[stid] = 0;
                       atomicAdd(&stashcount,1);
                   //    atomicSub(&pathcount,1);
                       break; 
                   } 
                   level--; 
                   if (level<0) break;   
                   treeindex = (treeindex-1)/2;
                    
                   
                }		       

			 } 


         }*/
		// else if(tid<896&&tid>511) {          //other threads bring in the data from tree to data stash. 
           if (tid< 384){
              //int stid = tid - STASHSIZE*BLOCKPERBUCKET; 
              int bucketid= tid/16;
             int treeindex = calcindex(bucketid/2, pathid);
              int whichdata = tid%16;
              int whichblock = bucketid%2;
			  volatile int gabarge;
			    if(stashaccessloc[bucketid] !=999){           //999 means this block in the tree path is empty
              datastash[(stashaccessloc[bucketid]%STASHSIZE)*16+whichdata] = datatree[treeindex].block[whichblock].data[whichdata]; 
				}else{                             // if block in the tree path is empty , read the block still, but write to a garbage(don't care) position 
					gabarge =  datatree[treeindex].block[whichblock].data[whichdata];
				}

         }  
         __syncthreads();

         // writeback data back from stash to tree 
         if (tid < 384){
           int bucketid = tid/16;
             int treeindex = calcindex(bucketid/2, pathid);
              int whichdata = tid%16;
              int whichblock = bucketid%2;
			
             datatree[treeindex].block[whichblock].data[whichdata] = datastash[(writebackloc[bucketid]%STASHSIZE)*16+whichdata];       
             //if (writebackloc[bucketid] >=STASHSIZE){
             //   printf("invalllllllll\n %d access, %d thread", i,tid );
            // }

         }else if (tid< 400){
             int stid = tid-384;
	     checktable2[i].data[stid] = datastash[expectedblockindex*16+stid];   

         }  
	//	 __syncthreads();
	
     
	}

    

    //if (tid == 0)    printf("max stash size %ud\n",256-maxstashcount);
}
__global__ void setup_kernel(hiprandState *state)
{
int id = threadIdx.x;
/* Each thread gets same seed, a different sequence number,
no offset */
hiprand_init(1234, id, 0, &state[id]);
}


int main(int argc, char** argv)
{
    // Initialize command line
	 hipDeviceSetCacheConfig(hipFuncCachePreferShared);
//	 hipDeviceSetCacheConfig(hipFuncCachePreferL1);
      
    printf("start\n");
   hiprandState *devStates;
   CUDA_CALL(hipMalloc((void **)&devStates, 1024 * sizeof(hiprandState)));
   setup_kernel<<<CUDABLOCKNUM,CUDATHREADNUM>>>(devStates);
    uint16_t* p_table = new uint16_t[1<<BLOCKNUMLOG];
    uint16_t* check_table = new uint16_t[ACCESSNUM];
    TDBlock<BLOCKSIZE>* check_table2 = new TDBlock<BLOCKSIZE>[ACCESSNUM];
	TDBlock<BLOCKSIZE>* resultlist = new TDBlock<BLOCKSIZE>[1<<BLOCKNUMLOG];
    uint32_t* access_script = new uint32_t[ACCESSNUM];
    //uint32_t* orampath = new uint32_t[1<<LEAFNUMLOG];
    OramB* oramtree = new OramB[TREESIZE];
    OramD* doramtree = new OramD[TREESIZE];

    printf("sizeof OramD %d\n", sizeof(OramD));
    for (int i = 0; i< (1<<(BLOCKNUMLOG)); i++){
        p_table[i] = 0xdead;
    }
    printf("finished initialize raw p_table\n");
    int startpoint =  (1<<LEAFNUMLOG) -1 ; 
    for (int i =0; i< startpoint;i++){
	oramtree[i].initzero();

    }
    for (int i = startpoint; i< (TREESIZE-1); i++){
            int temp = i- startpoint;
        doramtree[i].init(); 
        oramtree[i].init(temp*BLOCKPERBUCKET); 
        p_table[BLOCKPERBUCKET*temp] = temp;
        p_table[BLOCKPERBUCKET*temp+1] = temp;
        resultlist[BLOCKPERBUCKET*temp] =doramtree[i].block[0];
		resultlist[BLOCKPERBUCKET*temp+1] =doramtree[i].block[1];

        //oramtree[i].id[0] =( 0x8000 |rand()%(1<<BLOCKNUMLOG)); 
        //p_table[oramtree[i].id[0]%(1<<BLOCKNUMLOG)] = temp; 
        //oramtree[i].id[1] =( 0x8000| rand()%(1<<BLOCKNUMLOG)); 
        //p_table[oramtree[i].id[1]%(1<<BLOCKNUMLOG)] = temp; 
    }
    printf ("finished initializa p_table \n");
    printf ("Accessing %d blocks \n", ACCESSNUM);

    for (int i = 0; i<(1<BLOCKNUMLOG); i++){
	if (p_table[i] != i/BLOCKPERBUCKET)  printf("p_table is wrong\n");

      printf("finish checking position table\n");
    }
   
    for (int i = 0; i<(ACCESSNUM); i++){
        access_script[i] = rand()%(1<<BLOCKNUMLOG);
          if (p_table[access_script[i]] == 0xdead) printf("p_table has hole?\n");

        for(int j =0; j<16; j++){
        check_table2[i].data[j]=0xdeadbeef;
        }
       // printf("host access : 0x%x\n", p_table[access_script[i]] );
    }
    printf("finish initialing host\n");
    printf("orambucket size %d \n",sizeof(OramB));
    uint16_t* cup_table;
    uint16_t* cucheck_table;
    TDBlock<BLOCKSIZE>* cucheck_table2;
    uint32_t* cuaccess_script;
   // uint32_t* cuorampath;
    OramB* cuoramtree;
    OramD* cudoramtree; 
    hipError_t pterr = hipMalloc((void**)&cup_table,sizeof(uint16_t) *( 1<<BLOCKNUMLOG));
    if(pterr != hipSuccess){
     printf("The pterror is %s", hipGetErrorString(pterr));
    }
    hipError_t err = hipMalloc((void**)&cucheck_table,sizeof(uint16_t)*(ACCESSNUM));
    if(err != hipSuccess){
     printf("The error is %s", hipGetErrorString(err));
    }
    hipError_t errr = hipMalloc((void**)&cucheck_table2,sizeof(TDBlock<BLOCKSIZE>)*(ACCESSNUM));
    if(errr != hipSuccess){
     printf("The error2 is %s", hipGetErrorString(errr));
    }
    hipMalloc((void**)&cuaccess_script,sizeof(uint32_t) *(ACCESSNUM));
   // hipMalloc((void**)&cuorampath,sizeof(uint32_t) *( 1<<LEAFNUMLOG));
    hipMalloc((void**)&cuoramtree,sizeof(OramB) *( TREESIZE));
    hipMalloc((void**)&cudoramtree,sizeof(OramD) *( TREESIZE));
    
    hipError_t pterr2 = hipMemcpy(cup_table, p_table, (1<<BLOCKNUMLOG) * sizeof(uint16_t),hipMemcpyHostToDevice);
    if(pterr2 != hipSuccess){
     printf("The pt copy htom error is %s", hipGetErrorString(pterr2));
    }
   
    hipMemcpy(cuaccess_script, access_script, (ACCESSNUM) * sizeof(uint32_t),hipMemcpyHostToDevice);
   // hipMemcpy(cuorampath, orampath, (1<<LEAFNUMLOG) * sizeof(uint32_t),hipMemcpyHostToDevice);
    hipMemcpy(cuoramtree, oramtree, (TREESIZE) * sizeof(OramB),hipMemcpyHostToDevice);
    hipMemcpy(cudoramtree, doramtree, (TREESIZE) * sizeof(OramD),hipMemcpyHostToDevice);
    CUDA_CALL(hipMemcpy(cucheck_table2, check_table2,(ACCESSNUM)*sizeof(TDBlock<BLOCKSIZE>), hipMemcpyHostToDevice));
    oramshare<<<CUDABLOCKNUM,CUDATHREADNUM>>>(cup_table,cuaccess_script,cucheck_table, cuoramtree, cucheck_table2, devStates,cudoramtree);
    if (hipPeekAtLastError() != hipSuccess) {
    	printf("The peek last error is %s", hipGetErrorString(hipGetLastError()));
    }
    hipDeviceSynchronize();
   // CUDA_CALL(hipMemcpy(doramtree,cudoramtree,(TREESIZE)*sizeof(OramD),hipMemcpyDeviceToHost));
    hipError_t err2 = hipMemcpy(check_table, cucheck_table, (ACCESSNUM) * sizeof(uint16_t), hipMemcpyDeviceToHost);
    if(err2 != hipSuccess){
     printf("after  checktable copy error is %s\n", hipGetErrorString(err2));
    }
    hipError_t err3 = hipMemcpy(check_table2, cucheck_table2, (ACCESSNUM) * sizeof(TDBlock<BLOCKSIZE>), hipMemcpyDeviceToHost);
    if(err3 != hipSuccess){
     printf("after  checktable copy error is %s\n", hipGetErrorString(err3));
    }
    printf("gpu finished\n");
    bool pass = true; 
	bool dpass = true; 
    for (int i =0 ; i< ACCESSNUM ; i++){
		if (check_table[i] != access_script[i]){
		//	printf("fail test, access number: %d\n", i);
			pass = false; 
		}

		if( check_table2[i] != resultlist[access_script[i]]){
			dpass = false;
                       printf("access data %d not correct\n",i);
                      for(int j = 0; j< 16;j++){
                      printf("expected: %x, actual %x\n",resultlist[access_script[i]].data[j], check_table2[i].data[j]);
                      }
		}
      // int bucketindex = (1<<LEAFNUMLOG) - 1 + p_table[access_script[i]]; 
       //printf ("bucket index %d \n", bucketindex);
       /*for(int j = 11 ; j >= 0; j--) {
         
        if (check_table[i*24+j*2] !=  oramtree[bucketindex].id[0]){
            pass = false; 
            printf("fail 0 id: 0x%x 0x%x real id 0x%X,  0x%x\n" ,bucketindex,oramtree[bucketindex].id[0],check_table2[i*24+j*2], check_table[i*24+j*2] );
        }
        else
        { 
            printf("pass 0 id: 0x%x 0x%x real id 0x%x, 0x%x\n" ,bucketindex,oramtree[bucketindex].id[0],check_table2[i*24+j*2] ,check_table[i*24+j*2] );
        }
        if (check_table[i*24+j*2+1] !=  oramtree[bucketindex].id[1]){
            pass = false;
            printf("fail 1 id: 0x%x 0x%x real id 0x%x 0x%x\n" ,bucketindex, oramtree[bucketindex].id[1], check_table2[i*24+j*2+1],check_table[i*24+j*2+1] );
           
        }
        else
        { 
            printf("pass 0 id: 0x%x 0x%x real id 0x%x 0x%x\n" ,bucketindex,oramtree[bucketindex].id[0],check_table2[i*24+j*2+1] ,check_table[i*24+j*2] );
        }
         bucketindex = (bucketindex-1)/2; 

       }*/
	}
    
    printf("\nfinished \n");
    if (pass) {
		printf("All meta data correct\n");
	}else{
		printf("Some meta data not correct\n");
	}

	 if (dpass) {
		printf("All data correct\n");
	}else{
		printf("Some data not correct\n");
	}
    
    hipFree(cuaccess_script);
    hipFree(cup_table);
    hipFree(cucheck_table);
    hipFree(cucheck_table2);
    //hipFree(cuorampath);
    hipFree(cuoramtree);
    hipFree(cudoramtree);
    hipFree(devStates);
    delete[] p_table;
    delete[] access_script;
    //delete[] orampath;
    delete[] check_table; 
    delete[] check_table2; 
    delete[] oramtree; 
    delete[] doramtree; 
	delete[] resultlist; 
    hipDeviceReset();
    return 0;
}
